#include "CuNeuralNetwork.h"

#include <iostream>
#include <sstream>
#include <numeric>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>
#include <FreeImage.h>
#include <unistd.h>
#include <string.h>
#include <algorithm>

//////////////////////////////////////////////////////////////////////////////
// Error handling
// Adapted from the CUDNN classification code
// sample: https://developer.nvidia.com/cuDNN

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

void CuNeuralNetwork::initializeConvolutionalLayerTensorDescriptor(
		hipdnnHandle_t *cudnnHandle,
		hipdnnTensorDescriptor_t * inputDataTensorDescriptor,
		hipdnnFilterDescriptor_t * kernelDescriptor,
		hipdnnConvolutionDescriptor_t * convolutionDescriptor,
		hipdnnTensorDescriptor_t * outputDataTensorDescriptor,
		hipdnnConvolutionFwdAlgo_t * algorithm, int executeBatchSize,
		int imageHeight, int imageWidth, int kernelHeight, int kernelWidth,
		int inputFeaturemaps, int outputFeaturemaps,
		size_t * workspaceSizeInByte, int * outputImages,
		int * outputFeaturemapsForEachImage, int * outputFeaturemapHeight,
		int * outputFeaturemapWidth) {

	//输入数据设定
	checkCUDNN(hipdnnCreateTensorDescriptor(inputDataTensorDescriptor));
	checkCUDNN(
			hipdnnSetTensor4dDescriptor(*inputDataTensorDescriptor,
					HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, executeBatchSize,
					inputFeaturemaps, imageHeight, imageWidth));

	//卷积核设定
	checkCUDNN(hipdnnCreateFilterDescriptor(kernelDescriptor));
	checkCUDNN(
			hipdnnSetFilter4dDescriptor(*kernelDescriptor, HIPDNN_DATA_FLOAT,
					outputFeaturemaps, inputFeaturemaps, kernelHeight,
					kernelWidth));

	//卷积操作设定
	checkCUDNN(hipdnnCreateConvolutionDescriptor(convolutionDescriptor));
	//零填充的行数与列数：0 卷积的水平和垂直的滑动长度：1 x，y向上取样的比例尺：1
	//不使用卷积操作，因为卷积操作要旋转卷积核，而互相关操作无需旋转卷积核
	checkCUDNN(
			hipdnnSetConvolution2dDescriptor(*convolutionDescriptor, 0, 0, 1, 1,
					1, 1, HIPDNN_CROSS_CORRELATION));

	//输出数据设定
	//获取：图片数量，输出featuremap数量，featuremap的高度，featuremap的宽度

	checkCUDNN(
			hipdnnGetConvolution2dForwardOutputDim(*convolutionDescriptor,
					*inputDataTensorDescriptor, *kernelDescriptor, outputImages,
					outputFeaturemapsForEachImage, outputFeaturemapHeight,
					outputFeaturemapWidth));

	checkCUDNN(hipdnnCreateTensorDescriptor(outputDataTensorDescriptor));
	checkCUDNN(
			hipdnnSetTensor4dDescriptor(*outputDataTensorDescriptor,
					HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, *outputImages,
					*outputFeaturemapsForEachImage, *outputFeaturemapHeight,
					*outputFeaturemapWidth));

	//选择FP算法
	checkCUDNN(
			hipdnnGetConvolutionForwardAlgorithm(*cudnnHandle,
					*inputDataTensorDescriptor, *kernelDescriptor,
					*convolutionDescriptor, *outputDataTensorDescriptor,
					HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, algorithm));

	//获取workspace的大小
	checkCUDNN(
			hipdnnGetConvolutionForwardWorkspaceSize(*cudnnHandle,
					*inputDataTensorDescriptor, *kernelDescriptor,
					*convolutionDescriptor, *outputDataTensorDescriptor,
					*algorithm, workspaceSizeInByte));

}

void CuNeuralNetwork::syncTrainingDataToDevice(float * h_input_data,
		float * d_input_data, float * h_kernel, float * d_kernel,
		float * d_output_data, void * d_cudnn_workspace, int executeBatchSize,
		int imageHeight, int imageWidth, int kernelHeight, int kernelWidth,
		int inputFeaturemaps, int outputFeaturemaps, int outputImages,
		int outputFeaturemapsForEachImage, int outputFeaturemapHeight,
		int outputFeaturemapWidth, size_t workspaceSizeInByte) {

	//输入层数据同步
	checkCudaErrors(
			hipMalloc(&d_input_data,
					sizeof(float) * executeBatchSize * inputFeaturemaps
							* imageHeight * imageWidth));
	checkCudaErrors(
			hipMemcpyAsync(d_input_data, h_input_data,
					sizeof(float) * executeBatchSize * inputFeaturemaps
							* imageHeight * imageWidth,
					hipMemcpyHostToDevice));

	//卷积核数据同步
	checkCudaErrors(
			hipMalloc(&d_kernel,
					sizeof(float) * outputFeaturemaps * kernelHeight
							* kernelWidth));
	checkCudaErrors(
			hipMemcpyAsync(d_kernel, h_kernel,
					sizeof(float) * outputFeaturemaps * kernelHeight
							* kernelWidth, hipMemcpyHostToDevice));

	//输出层数据同步
	//输出层只做空间分配，不做数据同步

	checkCudaErrors(
			hipMalloc(&d_output_data,
					sizeof(float) * outputImages * outputFeaturemapsForEachImage
							* outputFeaturemapHeight * outputFeaturemapWidth));

	std::cout << outputImages << std::endl;

	//workspace空间分配
	checkCudaErrors(hipMalloc(&d_cudnn_workspace, workspaceSizeInByte));

	checkCudaErrors(hipDeviceSynchronize());

}
