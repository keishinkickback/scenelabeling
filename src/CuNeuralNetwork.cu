#include "includes/CuNeuralNetwork.h"

#include <iostream>
#include <sstream>
#include <numeric>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>
#include <FreeImage.h>
#include <unistd.h>
#include <string.h>
#include <algorithm>

//////////////////////////////////////////////////////////////////////////////
// Error handling
// Adapted from the CUDNN classification code
// sample: https://developer.nvidia.com/cuDNN

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

float * CuNeuralNetwork::initializeInputDataLayer(float * h_input_data,
		hipdnnTensorDescriptor_t * inputDataTensorDescriptor, int batchSize,
		int inputFeaturemaps, int imageHeight, int imageWidth) {

	checkCUDNN(hipdnnCreateTensorDescriptor(inputDataTensorDescriptor));
	checkCUDNN(
			hipdnnSetTensor4dDescriptor(*inputDataTensorDescriptor,
					HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize,
					inputFeaturemaps, imageHeight, imageWidth));

	float * d_input_data;

	checkCudaErrors(
			hipMalloc(&d_input_data,
					sizeof(float) * batchSize * inputFeaturemaps * imageHeight
							* imageWidth));

	checkCudaErrors(
			hipMemcpyAsync(d_input_data, h_input_data,
					sizeof(float) * batchSize * inputFeaturemaps * imageHeight
							* imageWidth, hipMemcpyHostToDevice));

	return d_input_data;

}

float * CuNeuralNetwork::initializeKernels(float * h_kernel,
		hipdnnFilterDescriptor_t * kernelDescriptor, int inputFeaturemaps,
		int outputFeaturemaps, int kernelHeight, int kernelWidth) {

	checkCUDNN(hipdnnCreateFilterDescriptor(kernelDescriptor));
	checkCUDNN(
			hipdnnSetFilter4dDescriptor(*kernelDescriptor, HIPDNN_DATA_FLOAT,
					outputFeaturemaps, inputFeaturemaps, kernelHeight,
					kernelWidth));
	float *d_kernel;
	checkCudaErrors(
			hipMalloc(&d_kernel,
					sizeof(float) * outputFeaturemaps * inputFeaturemaps
							* kernelHeight * kernelWidth));
	checkCudaErrors(
			hipMemcpyAsync(d_kernel, h_kernel,
					sizeof(float) * outputFeaturemaps * inputFeaturemaps
							* kernelHeight * kernelWidth,
					hipMemcpyHostToDevice));
	return d_kernel;

}

float * CuNeuralNetwork::initializeOutputDataLayer(
		hipdnnTensorDescriptor_t * inputDataTensorDescriptor,
		hipdnnFilterDescriptor_t * kernelDescriptor,
		hipdnnConvolutionDescriptor_t * convolutionDescriptor,
		hipdnnTensorDescriptor_t * outputDataTensorDescriptor,
		cudnnOutputDim * outputDim) {

	checkCUDNN(
			hipdnnGetConvolution2dForwardOutputDim(*convolutionDescriptor,
					*inputDataTensorDescriptor, *kernelDescriptor,
					&outputDim->outputImages,
					&outputDim->outputFeaturemapsForEachImage,
					&outputDim->outputFeaturemapHeight,
					&outputDim->outputFeaturemapWidth));

	checkCUDNN(hipdnnCreateTensorDescriptor(outputDataTensorDescriptor));
	checkCUDNN(
			hipdnnSetTensor4dDescriptor(*outputDataTensorDescriptor,
					HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
					outputDim->outputImages,
					outputDim->outputFeaturemapsForEachImage,
					outputDim->outputFeaturemapHeight,
					outputDim->outputFeaturemapWidth));
	//device上分配内存空间
	float *d_output_data;
	checkCudaErrors(
			hipMalloc(&d_output_data,
					sizeof(float) * outputDim->outputImages
							* outputDim->outputFeaturemapsForEachImage
							* outputDim->outputFeaturemapHeight
							* outputDim->outputFeaturemapWidth));

	return d_output_data;

}

float * CuNeuralNetwork::addBiasUnits(float * h_bias,
		hipdnnTensorDescriptor_t * biasTensorDescriptor, int outputFeaturemaps,
		int kernelHeight, int kernelWidth) {

	//偏置项设定
	checkCUDNN(hipdnnCreateTensorDescriptor(biasTensorDescriptor));
	checkCUDNN(
			hipdnnSetTensor4dDescriptor(*biasTensorDescriptor, HIPDNN_TENSOR_NCHW,
					HIPDNN_DATA_FLOAT, 1, outputFeaturemaps, 1, 1));

	//分配内存
	float *d_bias;
	checkCudaErrors(
			hipMalloc(&d_bias,
					sizeof(float) * outputFeaturemaps * kernelHeight
							* kernelWidth));
	checkCudaErrors(
			hipMemcpyAsync(d_bias, h_bias,
					sizeof(float) * outputFeaturemaps * kernelHeight
							* kernelWidth, hipMemcpyHostToDevice));

	return d_bias;

}

float * CuNeuralNetwork::initializePoolingLayer(float * d_output_data,
		hipdnnTensorDescriptor_t * inputDataTensorDescriptor,
		hipdnnPoolingDescriptor_t * poolingDescriptor,
		hipdnnTensorDescriptor_t * poolingDataTensorDescriptor,
		cudnnOutputDim * outputDim, int poolingWindowHeight,
		int poolingWindowWidth, int poolingVerticalStride,
		int poolingHorizontalStride, cudnnOutputDim * poolingOutputDim) {

	float * d_pooling_output_data;
	int poolingOutputDataHeight = (outputDim->outputFeaturemapHeight
			- poolingWindowHeight) / poolingHorizontalStride + 1;
	int poolingOutputDataWidth = (outputDim->outputFeaturemapWidth
			- poolingWindowWidth) / poolingVerticalStride + 1;

	checkCudaErrors(
			hipMalloc(&d_pooling_output_data,
					sizeof(float) * outputDim->outputImages
							* outputDim->outputFeaturemapsForEachImage
							* poolingOutputDataHeight
							* poolingOutputDataWidth));

	checkCUDNN(hipdnnCreatePoolingDescriptor(poolingDescriptor));
	checkCUDNN(
			hipdnnSetPooling2dDescriptor(*poolingDescriptor, HIPDNN_POOLING_MAX,
					poolingWindowHeight, poolingWindowWidth, 0, 0,
					poolingVerticalStride, poolingHorizontalStride));

	//没有实装这个方法？
//	checkCUDNN(
//			hipdnnGetPooling2dForwardOutputDim(
//					*poolingDescriptor,
//					*inputDataTensorDescriptor,
//					&poolingOutputDim->outputImages,
//					&poolingOutputDim->outputFeaturemapsForEachImage,
//					&poolingOutputDim->outputFeaturemapHeight,
//					&poolingOutputDim->outputFeaturemapWidth));
//
//	std::cout << poolingOutputDim->outputImages << " "
//			<< poolingOutputDim->outputFeaturemapsForEachImage << " "
//			<< poolingOutputDim->outputFeaturemapHeight << " "
//			<< poolingOutputDim->outputFeaturemapWidth << std::endl;

	poolingOutputDim->outputImages = outputDim->outputImages;
	poolingOutputDim->outputFeaturemapsForEachImage =
			outputDim->outputFeaturemapsForEachImage;
	poolingOutputDim->outputFeaturemapHeight = poolingOutputDataHeight;
	poolingOutputDim->outputFeaturemapWidth = poolingOutputDataWidth;

	checkCUDNN(hipdnnCreateTensorDescriptor(poolingDataTensorDescriptor));
	checkCUDNN(
			hipdnnSetTensor4dDescriptor(*poolingDataTensorDescriptor,
					HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
					outputDim->outputImages,
					outputDim->outputFeaturemapsForEachImage,
					poolingOutputDataHeight, poolingOutputDataWidth));

	return d_pooling_output_data;

}

float * computeFullconnectedDataLayer(
		hipblasHandle_t cublasHandle,
		float * d_input_data,
		float * d_kernel,
		float * d_bias,
		float * d_ones_vector,
		int outputFeaturemapLength,
		int inputFeaturemaps,
		int inputFeaturemapHeight, int inputFeaturemapWidth) {

	float * d_output_data;

//	int featuremapsLength = inputFeaturemapHeight * inputFeaturemapWidth
//			* inputFeaturemaps;
//
//	checkCudaErrors(
//			hipMalloc((void** ) &d_output_data,
//					outputFeaturemaps * sizeof(float)));
//
//	checkCudaErrors(
//			hipMemset(d_output_data, 0,
//					outputFeaturemapLength * sizeof(float)));
//
//	float alpha = 1.0;
//	float beta = 0;
//
//	//全连接运算
//	checkCudaErrors(
//			hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, outputFeaturemaps, 1, featuremapsLength, &alpha, d_input_data, featuremapsLength, d_kernel, featuremapsLength, &beta, d_output_data, outputFeaturemaps));
//
//	//加上偏置项
//	//d_ones_vector应为1*1
//	beta = 1;
//	checkCudaErrors(
//			hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, outputFeaturemaps, 1, 1, &alpha, d_bias, featuremapsLength, d_ones_vector, featuremapsLength, &beta, d_output_data, outputFeaturemaps));

	return d_output_data;

}
