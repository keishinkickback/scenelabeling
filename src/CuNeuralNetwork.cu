#include "CuNeuralNetwork.h"

#include <iostream>
#include <sstream>
#include <numeric>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>
#include <FreeImage.h>
#include <unistd.h>
#include <string.h>
#include <algorithm>

//////////////////////////////////////////////////////////////////////////////
// Error handling
// Adapted from the CUDNN classification code
// sample: https://developer.nvidia.com/cuDNN

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

float * CuNeuralNetwork::createInputDataLayer(float * h_input_data,
		hipdnnTensorDescriptor_t * inputDataTensorDescriptor, int batchSize,
		int inputFeaturemaps, int imageHeight, int imageWidth) {

	checkCUDNN(hipdnnCreateTensorDescriptor(inputDataTensorDescriptor));
	checkCUDNN(
			hipdnnSetTensor4dDescriptor(*inputDataTensorDescriptor,
					HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize,
					inputFeaturemaps, imageHeight, imageWidth));

	float * d_input_data;

	checkCudaErrors(
			hipMalloc(&d_input_data,
					sizeof(float) * batchSize * inputFeaturemaps * imageHeight
							* imageWidth));

	checkCudaErrors(
			hipMemcpyAsync(d_input_data, h_input_data,
					sizeof(float) * batchSize * inputFeaturemaps * imageHeight
							* imageWidth, hipMemcpyHostToDevice));

	return d_input_data;

}

float * CuNeuralNetwork::createKernel(float * h_kernel,
		hipdnnFilterDescriptor_t * kernelDescriptor, int inputFeaturemaps,
		int outputFeaturemaps, int kernelHeight, int kernelWidth) {

	checkCUDNN(hipdnnCreateFilterDescriptor(kernelDescriptor));
	checkCUDNN(
			hipdnnSetFilter4dDescriptor(*kernelDescriptor, HIPDNN_DATA_FLOAT,
					outputFeaturemaps, inputFeaturemaps, kernelHeight,
					kernelWidth));
	float *d_kernel;
	checkCudaErrors(
			hipMalloc(&d_kernel,
					sizeof(float) * outputFeaturemaps * kernelHeight
							* kernelWidth));
	checkCudaErrors(
			hipMemcpyAsync(d_kernel, h_kernel,
					sizeof(float) * outputFeaturemaps * kernelHeight
							* kernelWidth, hipMemcpyHostToDevice));
	return d_kernel;

}

float * CuNeuralNetwork::createOutputDataLayer(
		hipdnnTensorDescriptor_t * inputDataTensorDescriptor,
		hipdnnFilterDescriptor_t * kernelDescriptor,
		hipdnnConvolutionDescriptor_t * convolutionDescriptor,
		hipdnnTensorDescriptor_t * outputDataTensorDescriptor,
		OutputDim * outputDim) {

	checkCUDNN(
			hipdnnGetConvolution2dForwardOutputDim(*convolutionDescriptor,
					*inputDataTensorDescriptor, *kernelDescriptor,
					&outputDim->outputImages,
					&outputDim->outputFeaturemapsForEachImage,
					&outputDim->outputFeaturemapHeight,
					&outputDim->outputFeaturemapWidth));

	checkCUDNN(hipdnnCreateTensorDescriptor(outputDataTensorDescriptor));
	checkCUDNN(
			hipdnnSetTensor4dDescriptor(*outputDataTensorDescriptor,
					HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
					outputDim->outputImages,
					outputDim->outputFeaturemapsForEachImage,
					outputDim->outputFeaturemapHeight,
					outputDim->outputFeaturemapWidth));
	//device上分配内存空间
	float *d_output_data;
	checkCudaErrors(
			hipMalloc(&d_output_data,
					sizeof(float) * outputDim->outputImages
							* outputDim->outputFeaturemapsForEachImage
							* outputDim->outputFeaturemapHeight
							* outputDim->outputFeaturemapWidth));

	return d_output_data;

}

float * CuNeuralNetwork::addBiasUnits(float * h_bias,
		hipdnnTensorDescriptor_t * biasTensorDescriptor, int outputFeaturemaps,
		int kernelHeight, int kernelWidth) {

	//偏置项设定
	checkCUDNN(hipdnnCreateTensorDescriptor(biasTensorDescriptor));
	checkCUDNN(
			hipdnnSetTensor4dDescriptor(*biasTensorDescriptor, HIPDNN_TENSOR_NCHW,
					HIPDNN_DATA_FLOAT, 1, outputFeaturemaps, 1, 1));

	//分配内存
	float *d_bias;
	checkCudaErrors(
			hipMalloc(&d_bias,
					sizeof(float) * outputFeaturemaps * kernelHeight
							* kernelWidth));
	checkCudaErrors(
			hipMemcpyAsync(d_bias, h_bias,
					sizeof(float) * outputFeaturemaps * kernelHeight
							* kernelWidth, hipMemcpyHostToDevice));

	return d_bias;

}

