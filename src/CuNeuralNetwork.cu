#include "CuNeuralNetwork.h"

#include <iostream>
#include <sstream>
#include <numeric>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>
#include <FreeImage.h>
#include <unistd.h>
#include <string.h>
#include <algorithm>

//////////////////////////////////////////////////////////////////////////////
// Error handling
// Adapted from the CUDNN classification code
// sample: https://developer.nvidia.com/cuDNN

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

void CuNeuralNetwork::initializeConvolutionalLayerTensorDescriptor(
		hipdnnHandle_t *cudnnHandle,
		hipdnnTensorDescriptor_t * inputDataTensorDescriptor,
		hipdnnFilterDescriptor_t * kernelDescriptor,
		hipdnnConvolutionDescriptor_t * convolutionDescriptor,
		hipdnnTensorDescriptor_t * outputDataTensorDescriptor,
		hipdnnConvolutionFwdAlgo_t * algorithm, int executeBatchSize,
		int imageHeight, int imageWidth, int kernelHeight, int kernelWidth,
		int inputFeaturemaps, int outputFeaturemaps,
		size_t * workspaceSizeInByte, int * outputImages,
		int * outputFeaturemapsForEachImage, int * outputFeaturemapHeight,
		int * outputFeaturemapWidth) {

	//输入数据设定
	checkCUDNN(hipdnnCreateTensorDescriptor(inputDataTensorDescriptor));
	checkCUDNN(
			hipdnnSetTensor4dDescriptor(*inputDataTensorDescriptor,
					HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, executeBatchSize,
					inputFeaturemaps, imageHeight, imageWidth));

	//卷积核设定
	checkCUDNN(hipdnnCreateFilterDescriptor(kernelDescriptor));
	checkCUDNN(
			hipdnnSetFilter4dDescriptor(*kernelDescriptor, HIPDNN_DATA_FLOAT,
					outputFeaturemaps, inputFeaturemaps, kernelHeight,
					kernelWidth));

	//卷积操作设定
	checkCUDNN(hipdnnCreateConvolutionDescriptor(convolutionDescriptor));
	//零填充的行数与列数：0 卷积的水平和垂直的滑动长度：1 x，y向上取样的比例尺：1
	//不使用卷积操作，因为卷积操作要旋转卷积核，而互相关操作无需旋转卷积核
	checkCUDNN(
			hipdnnSetConvolution2dDescriptor(*convolutionDescriptor, 0, 0, 1, 1,
					1, 1, HIPDNN_CROSS_CORRELATION));

	//输出数据设定
	//获取：图片数量，输出featuremap数量，featuremap的高度，featuremap的宽度

	checkCUDNN(
			hipdnnGetConvolution2dForwardOutputDim(*convolutionDescriptor,
					*inputDataTensorDescriptor, *kernelDescriptor, outputImages,
					outputFeaturemapsForEachImage, outputFeaturemapHeight,
					outputFeaturemapWidth));

	checkCUDNN(hipdnnCreateTensorDescriptor(outputDataTensorDescriptor));
	checkCUDNN(
			hipdnnSetTensor4dDescriptor(*outputDataTensorDescriptor,
					HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, *outputImages,
					*outputFeaturemapsForEachImage, *outputFeaturemapHeight,
					*outputFeaturemapWidth));

	//选择FP算法
	checkCUDNN(
			hipdnnGetConvolutionForwardAlgorithm(*cudnnHandle,
					*inputDataTensorDescriptor, *kernelDescriptor,
					*convolutionDescriptor, *outputDataTensorDescriptor,
					HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, algorithm));

	//获取workspace的大小
	checkCUDNN(
			hipdnnGetConvolutionForwardWorkspaceSize(*cudnnHandle,
					*inputDataTensorDescriptor, *kernelDescriptor,
					*convolutionDescriptor, *outputDataTensorDescriptor,
					*algorithm, workspaceSizeInByte));

}
