#include "hip/hip_runtime.h"
#include "KernelGenerator.h"

#include <iostream>
#include <sstream>
#include <numeric>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>
#include <FreeImage.h>
#include <unistd.h>
#include <string.h>
#include <algorithm>

void KernelGenerator::initializeKernelUsingXavierAlgorithm(int kernelHeight,
		int kernelWeight, int outputFeaturemaps, std::vector<float> * kernel) {

	//随机数生成器初始化
	std::random_device rd;
	//使用马特赛特旋转演算法伪随机数生成器
	std::mt19937 generator(rd());

	//Xavier算法,分母为kernel的输出维度
	//参考：http://caffe.berkeleyvision.org/doxygen/classcaffe_1_1XavierFiller.html
	float scale = sqrt(
			3.0f / (kernelHeight * kernelWeight * outputFeaturemaps));

	std::uniform_real_distribution<> distribution(-scale, scale);

	for (int i = 0; i < kernel->size(); i++) {
		kernel->at(i) = static_cast<float>(distribution(generator));
	}

}

void KernelGenerator::initializeBiasUsingXavierAlgorithm(int outputFeaturemaps,
		std::vector<float> * bias) {

	std::random_device rd;
	std::mt19937 generator(rd());

	float scale = sqrt(3.0f / (outputFeaturemaps));

	std::uniform_real_distribution<> distribution(-scale, scale);

	for (int i = 0; i < bias->size(); i++) {
		bias->at(i) = static_cast<float>(distribution(generator));
	}

}
