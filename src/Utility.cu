#include "hip/hip_runtime.h"
#include "Utility.h"

#include <iostream>
#include <sstream>
#include <numeric>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>
#include <FreeImage.h>
#include <unistd.h>
#include <string.h>
#include <algorithm>
#include <math.h>

float* Utility::VectorToArray(std::vector<float> * input) {

	float * array;
	array = new float[input->size()];

	for (int i = 0; i < input->size(); i++) {
		array[i] = input->at(i);
	}

	return array;
}

void Utility::PrintDynamicArray(float * array, int length) {
	for (int i = 0; i < length; i++) {
		std::cout << array[i] << std::endl;
	}
}

std::vector<std::vector<float> > Utility::ArrayToMatrix(float * array,
		int height, int width) {

	int arrayIndex = 0;
	std::vector<std::vector<float> > matrix;

	for (int y = 0; y < height; y++) {

		std::vector<float> row;

		for (int x = 0; x < width; x++) {

			row.push_back(array[arrayIndex]);
			arrayIndex++;

		}

		matrix.push_back(row);

	}

	return matrix;

}

bool Utility::floatIsEqual(float a, float b) {

	if (abs(a - b) <= 0.00001f) {

		return true;

	} else {

		return false;

	}

}
