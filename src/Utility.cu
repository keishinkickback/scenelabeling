#include "hip/hip_runtime.h"
#include "Utility.h"

#include <iostream>
#include <sstream>
#include <numeric>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>
#include <FreeImage.h>
#include <unistd.h>
#include <string.h>
#include <algorithm>
#include <math.h>

float* Utility::VectorToArray(std::vector<float> * input) {

	float * array;
	array = new float[input->size()];

	for (int i = 0; i < input->size(); i++) {
		array[i] = input->at(i);
	}

	return array;
}

void Utility::PrintDynamicArray(float * array, int length) {
	for (int i = 0; i < length; i++) {
		std::cout << array[i] << std::endl;
	}
}

std::vector<std::vector<float> > Utility::ArrayToMatrix(float * array,
		int height, int width) {

	int arrayIndex = 0;
	std::vector<std::vector<float> > matrix;

	for (int y = 0; y < height; y++) {

		std::vector<float> row;

		for (int x = 0; x < width; x++) {

			row.push_back(array[arrayIndex]);
			arrayIndex++;

		}

		matrix.push_back(row);

	}

	return matrix;

}

bool Utility::FloatIsEqual(float a, float b) {

	if (abs(a - b) <= 0.00001f) {

		return true;

	} else {

		return false;

	}

}

std::vector<float *> Utility::SplitArray(float * array, int part,
		int stepLength) {

	std::vector<float *> vector;

	for (int i = 1; i <= part; i++) {

		std::vector<float> subVector;

		for (int j = (i - 1) * stepLength; j < i * stepLength; j++) {

			subVector.push_back(array[j]);

		}

		vector.push_back(Utility::VectorToArray(&subVector));
	}

	return vector;

}
