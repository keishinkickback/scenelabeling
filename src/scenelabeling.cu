#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : scenelabeling.cu
 Author      : liujinhang @ whut
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <sstream>
#include <numeric>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>
#include <FreeImage.h>
#include <unistd.h>
#include <string.h>
#include <algorithm>

#include "CuNeuralNetwork.h"

#include <vector>

//////////////////////////////////////////////////////////////////////////////
// Error handling
// Adapted from the CUDNN classification code
// sample: https://developer.nvidia.com/cuDNN

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

class ImageProcessor {

public:
	void readRGBImage(char *imagePath, std::vector<float> *redChannel,
			std::vector<float> *greenChannel, std::vector<float> *blueChannel) {

		FreeImage_Initialise(TRUE);

		FIBITMAP* fib;
		fib = FreeImage_Load(FIF_PNG, imagePath, PNG_DEFAULT);
		int width = FreeImage_GetWidth(fib);
		int height = FreeImage_GetHeight(fib);

		RGBQUAD color;

		for (int x = 0; x < width; x++) {

			for (int y = 0; y < height; y++) {

				FreeImage_GetPixelColor(fib, x, y, &color);

				float blue = color.rgbBlue;
				float green = color.rgbGreen;
				float red = color.rgbRed;
				redChannel->push_back(red);
				greenChannel->push_back(green);
				blueChannel->push_back(blue);

			}

		}

		FreeImage_Unload(fib);
		FreeImage_DeInitialise();
	}

	std::vector<float> imageChannelNormalization(std::vector<float> *channel) {

		float maxColorChannel = *std::max_element(channel->begin(),
				channel->end());
		float minColorChannel = *std::min_element(channel->begin(),
				channel->end());

		std::vector<float> result;

		for (int i = 0; i < channel->size(); i++) {
			result.push_back(
					(channel->at(i) - minColorChannel)
							/ (maxColorChannel - minColorChannel));
		}

		channel->clear();

		return result;

	}
};

class KernelGenerator {

public:

	void initializeKernelUsingXavierAlgorithm(int kernelHeight,
			int kernelWeight, int channelNumber, std::vector<float> * kernel) {

		//随机数生成器初始化
		std::random_device rd;
		//使用马特赛特旋转演算法伪随机数生成器
		std::mt19937 generator(rd());

		float core = sqrt(3.0f / (kernelHeight * kernelWeight * channelNumber));

		std::uniform_real_distribution<> distribution(-core, core);

		for (int i = 0; i < kernel->size(); i++) {
			kernel->at(i) = static_cast<float>(distribution(generator));
		}

//		for(int i = 0 ; i < kernel->size();i++){
//			std::cout << kernel->at(i) << std::endl;
//		}

	}

};

//class CuNeuralNetwork {
//
//public:
//
//	//input
//	hipdnnHandle_t *cudnnHandle;
//	hipdnnTensorDescriptor_t * inputDataTensorDescriptor;
//	hipdnnFilterDescriptor_t * kernelDescriptor;
//	hipdnnConvolutionDescriptor_t * convolutionDescriptor;
//	hipdnnTensorDescriptor_t * outputDataTensorDescriptor;
//	int executeBatchSize;
//	int imageHeight;
//	int imageWidth;
//	int kernelHeight;
//	int kernelWidth;
//	int inputChannels;
//	int outputChannels;
//
//	//output
//	size_t workspaceSizeInByte;
//	hipdnnConvolutionFwdAlgo_t * algorithm;
//	int outputImageNumber;
//	int outputChannelsOfEachImage;
//	int outputFeaturemapHeight;
//	int outputFeaturemapWidth;
//
//public:
//
//	size_t initializeConvolutionalLayerTensorDescriptor(
//			hipdnnHandle_t *cudnnHandle,
//			hipdnnTensorDescriptor_t * inputDataTensorDescriptor,
//			hipdnnFilterDescriptor_t * kernelDescriptor,
//			hipdnnConvolutionDescriptor_t * convolutionDescriptor,
//			hipdnnTensorDescriptor_t * outputDataTensorDescriptor,
//			hipdnnConvolutionFwdAlgo_t * algorithm, int executeBatchSize,
//			int imageHeight, int imageWidth, int kernelHeight, int kernelWidth,
//			int inputChannels, int outputChannels) {
//
//		//输入数据设定
//		checkCUDNN(hipdnnCreateTensorDescriptor(inputDataTensorDescriptor));
//		checkCUDNN(
//				hipdnnSetTensor4dDescriptor(*inputDataTensorDescriptor,
//						HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, executeBatchSize,
//						inputChannels, imageHeight, imageWidth));
//
//		//卷积核设定
//		checkCUDNN(hipdnnCreateFilterDescriptor(kernelDescriptor));
//		checkCUDNN(
//				hipdnnSetFilter4dDescriptor(*kernelDescriptor, HIPDNN_DATA_FLOAT,
//						outputChannels, outputChannels, kernelHeight,
//						kernelWidth));
//
//		//卷积操作设定
//		checkCUDNN(hipdnnCreateConvolutionDescriptor(convolutionDescriptor));
//		//零填充的行数与列数：0 卷积的水平和垂直的滑动长度：1 x，y向上取样的比例尺：1
//		//不使用卷积操作，因为卷积操作要旋转卷积核，而互相关操作无需旋转卷积核
//		checkCUDNN(
//				hipdnnSetConvolution2dDescriptor(*convolutionDescriptor, 0, 0, 1,
//						1, 1, 1, HIPDNN_CROSS_CORRELATION));
//
//		//输出数据设定
//		//获取：图片数量，输出featuremap数量，featuremap的高度，featuremap的宽度
//
//		checkCUDNN(
//				hipdnnGetConvolution2dForwardOutputDim(*convolutionDescriptor,
//						*inputDataTensorDescriptor, *kernelDescriptor,
//						&this->imageNumber, &this->ChannelsOfImage,
//						&this->featuremapHeight, &this->featuremapWidth));
//
//		checkCUDNN(hipdnnCreateTensorDescriptor(outputDataTensorDescriptor));
//		checkCUDNN(
//				hipdnnSetTensor4dDescriptor(*outputDataTensorDescriptor,
//						HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
//						outputData->imageNumber, outputData->ChannelsOfImage,
//						outputData->featuremapHeight,
//						outputData->featuremapWidth));
//
//		//选择FP算法
//		checkCUDNN(
//				hipdnnGetConvolutionForwardAlgorithm(*cudnnHandle,
//						*inputDataTensorDescriptor, *kernelDescriptor,
//						*convolutionDescriptor, *outputDataTensorDescriptor,
//						HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, algorithm));
//
//		//获取workspace的大小
//		size_t workspaceSizeInByte = 0;
//		checkCUDNN(
//				hipdnnGetConvolutionForwardWorkspaceSize(*cudnnHandle,
//						*inputDataTensorDescriptor, *kernelDescriptor,
//						*convolutionDescriptor, *outputDataTensorDescriptor,
//						*algorithm, &workspaceSizeInByte));
//
//		return workspaceSizeInByte;
//
//	}
//
//};

class Utility {

public:
	static float* VectorToArray(std::vector<float> * input) {

		float * array;
		array = new float[input->size()];

		for (int i = 0; i < input->size(); i++) {
			array[i] = input->at(i);
		}

		return array;
	}
};

class TestCase {

public:
	static void TestCase1(float * data, float * kernel) {

		float sum = 0.0f;

		sum += data[0] * kernel[0];
		sum += data[1] * kernel[1];
		sum += data[2] * kernel[2];
		sum += data[540] * kernel[3];
		sum += data[541] * kernel[4];
		sum += data[542] * kernel[5];
		sum += data[1080] * kernel[6];
		sum += data[1081] * kernel[7];
		sum += data[1082] * kernel[8];

		std::cout << " Ground Truth : " << sum << std::endl;

	}

	static void printDynamicArray(float * array, int length) {
		for (int i = 0; i < length; i++) {
			std::cout << array[i] << std::endl;
		}
	}

};

int main() {

	int filePathMaxLength = 256;
	int imageHeight = 540;
	int imageWidth = 540;
	int kernelHeight = 3;
	int kernelWidth = 3;

	//读取图片到RGB三个通道
	char imagePath[filePathMaxLength];
	getcwd(imagePath, filePathMaxLength);
	strcat(imagePath, "/trainingset/");
	strcat(imagePath, "train1.png");

	std::vector<float> redChannel;
	std::vector<float> greenChannel;
	std::vector<float> blueChannel;

	ImageProcessor processor;
	processor.readRGBImage(imagePath, &redChannel, &greenChannel, &blueChannel);

	redChannel = processor.imageChannelNormalization(&redChannel);
	greenChannel = processor.imageChannelNormalization(&greenChannel);
	blueChannel = processor.imageChannelNormalization(&blueChannel);

	float * h_data;
	h_data = Utility::VectorToArray(&redChannel);
	//END

	//卷积核数据初始化
	KernelGenerator generator;
	std::vector<float> kernel(kernelHeight * kernelWidth);
	generator.initializeKernelUsingXavierAlgorithm(kernelHeight, kernelWidth, 1,
			&kernel);

	float * h_kernel;
	h_kernel = Utility::VectorToArray(&kernel);
	//END

	//GPU的查询与选择
	int GPUs;
	checkCudaErrors(hipGetDeviceCount(&GPUs));
	if (GPUs > 0) {
		checkCudaErrors(hipSetDevice(0));
	} else {
		return 0;
	}

	//cudnn初始化
	hipdnnHandle_t cudnnHandle = NULL;
	hipdnnCreate(&cudnnHandle);

	hipdnnTensorDescriptor_t inputDataTensor;
	hipdnnFilterDescriptor_t kernelDescriptor;
	hipdnnConvolutionDescriptor_t convolutionDescriptor;
	hipdnnTensorDescriptor_t outputDataTensor;
	hipdnnConvolutionFwdAlgo_t algorithm;
	int executeBatchSize = 1;
	int inputFeaturemaps = 1;
	int outputFeaturemaps = 1;
	size_t workspaceSizeInByte;
	int outputImages;
	int outputFeaturemapsForEachImage;
	int outputFeaturemapHeight;
	int outputFeaturemapWidth;

	CuNeuralNetwork network;

	network.initializeConvolutionalLayerTensorDescriptor(&cudnnHandle,
			&inputDataTensor, &kernelDescriptor, &convolutionDescriptor,
			&outputDataTensor, &algorithm, executeBatchSize, imageWidth,
			imageHeight, kernelHeight, kernelWidth, inputFeaturemaps,
			outputFeaturemaps, &workspaceSizeInByte, &outputImages,
			&outputFeaturemapsForEachImage, &outputFeaturemapHeight,
			&outputFeaturemapWidth);

	std::cout << outputImages << " " << outputFeaturemapsForEachImage << " "
			<< outputFeaturemapHeight << " " << outputFeaturemapWidth
			<< std::endl;

	//输入数据设定
//	hipdnnTensorDescriptor_t inputDataTensor;
//	checkCUDNN(hipdnnCreateTensorDescriptor(&inputDataTensor));
//	//第一层输入为图片原始数据，看做第一层的featuremap,数量为1
//	checkCUDNN(
//			hipdnnSetTensor4dDescriptor(inputDataTensor, HIPDNN_TENSOR_NCHW,
//					HIPDNN_DATA_FLOAT, 1, 1, imageHeight, imageWidth));

//device上分配内存空间
	float *d_data;
	checkCudaErrors(
			hipMalloc(&d_data,
					sizeof(float) * 1 * 1 * imageHeight * imageWidth));

	//拷贝数据到device
	checkCudaErrors(
			hipMemcpyAsync(d_data, h_data,
					sizeof(float) * 1 * 1 * imageHeight * imageWidth,
					hipMemcpyHostToDevice));

	//卷积核设定
	//输入featuremap为1个，即图片原始数据。输出为1个featuremap，所以需要1个卷积核。
//	int inputFeaturemapNumber = 1;
//	int outputFeaturemapNumber = 1;
//	hipdnnFilterDescriptor_t kernelDescriptor;
//	checkCUDNN(hipdnnCreateFilterDescriptor(&kernelDescriptor));
//	checkCUDNN(
//			hipdnnSetFilter4dDescriptor(kernelDescriptor, HIPDNN_DATA_FLOAT,
//					outputFeaturemapNumber, inputFeaturemapNumber, kernelHeight,
//					kernelWidth));
	//device上分配内存空间
	float *d_kernel;
	checkCudaErrors(
			hipMalloc(&d_kernel,
					sizeof(float) * 1 * 1 * kernelHeight * kernelWidth));
	//拷贝数据到device
	checkCudaErrors(
			hipMemcpyAsync(d_kernel, h_kernel,
					sizeof(float) * 1 * 1 * kernelHeight * kernelWidth,
					hipMemcpyHostToDevice));

//	//卷积操作设定
//	hipdnnConvolutionDescriptor_t convolutionDescriptor;
//	checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolutionDescriptor));
//	//零填充的行数与列数：0,卷积的水平和垂直的滑动长度：1,x，y向上取样的比例尺：1
//	//不使用卷积操作，因为卷积操作要旋转卷积核，而这里不需要旋转，互相关就是无需旋转的卷积乘法
//	checkCUDNN(
//			hipdnnSetConvolution2dDescriptor(convolutionDescriptor, 0, 0, 1, 1,
//					1, 1, HIPDNN_CROSS_CORRELATION));
//
//	//获取：图片数量，输出featuremap数量，featuremap的高度，featuremap的宽度
//	int outputN, outputC, outputH, outputW;
//	checkCUDNN(
//			hipdnnGetConvolution2dForwardOutputDim(convolutionDescriptor,
//					inputDataTensor, kernelDescriptor, &outputN, &outputC,
//					&outputH, &outputW));
//
//	//输出featuremap设定
//	hipdnnTensorDescriptor_t outputDataTensor;
//	checkCUDNN(hipdnnCreateTensorDescriptor(&outputDataTensor));
//	checkCUDNN(
//			hipdnnSetTensor4dDescriptor(outputDataTensor, HIPDNN_TENSOR_NCHW,
//					HIPDNN_DATA_FLOAT, outputN, outputC, outputH, outputW));
	//device上分配内存空间
	float *d_output_data;
//	checkCudaErrors(
//			hipMalloc(&d_output_data,
//					sizeof(float) * adapter.OutputData.imageNumber
//							* adapter.OutputData.ChannelsOfImage
//							* adapter.OutputData.featuremapHeight
//							* adapter.OutputData.featuremapWidth));

//	//选择fp算法
//	hipdnnConvolutionFwdAlgo_t algorithm;
//	checkCUDNN(
//			hipdnnGetConvolutionForwardAlgorithm(cudnnHandle, inputDataTensor,
//					kernelDescriptor, convolutionDescriptor, outputDataTensor,
//					HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algorithm));
//
//	//获取workspace的大小
//	size_t workspaceSizeInByte = 0;
//	checkCUDNN(
//			hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
//					inputDataTensor, kernelDescriptor, convolutionDescriptor,
//					outputDataTensor, algorithm, &workspaceSizeInByte));
	//device上分配内存空间
//void *d_cudnn_workspace = nullptr;
//																	checkCudaErrors(hipMalloc(&d_cudnn_workspace, workspaceSizeInByte));
//
//	checkCudaErrors(hipDeviceSynchronize());
//
//	//fp
//	float alpha = 1.0f, beta = 0.0f;
//
//	checkCUDNN(
//			hipdnnConvolutionForward(cudnnHandle, &alpha, inputDataTensor,
//					d_data, kernelDescriptor, d_kernel, convolutionDescriptor,
//					algorithm, d_cudnn_workspace, workspaceSizeInByte, &beta,
//					outputDataTensor, d_output_data));
//
//	//定义动态float数组
//	float * h_output_data = new float[outputN * outputC * outputH * outputH];
//
//	checkCudaErrors(
//			hipMemcpyAsync(h_output_data, d_output_data,
//					sizeof(float) * outputN * outputC * outputH * outputH,
//					hipMemcpyDeviceToHost));
//
//	checkCudaErrors(hipDeviceSynchronize());
//
//	TestCase::TestCase1(h_data, h_kernel);
//
//	std::cout << " test result : " << h_output_data[0] << std::endl;

	//checkCUDNN(hipdnnDestroyTensorDescriptor(redChannelDataTensor));

}

