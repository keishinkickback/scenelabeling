#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : scenelabeling.cu
 Author      : liujinhang @ whut
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <sstream>
#include <numeric>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>
#include <FreeImage.h>
#include <unistd.h>
#include <string.h>
#include <algorithm>

#include "CuNeuralNetwork.h"

#include <vector>

//////////////////////////////////////////////////////////////////////////////
// Error handling
// Adapted from the CUDNN classification code
// sample: https://developer.nvidia.com/cuDNN

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

class ImageProcessor {

public:
	void readRGBImage(char *imagePath, std::vector<float> *redChannel,
			std::vector<float> *greenChannel, std::vector<float> *blueChannel) {

		FreeImage_Initialise(TRUE);

		FIBITMAP* fib;
		fib = FreeImage_Load(FIF_PNG, imagePath, PNG_DEFAULT);
		int width = FreeImage_GetWidth(fib);
		int height = FreeImage_GetHeight(fib);

		RGBQUAD color;

		for (int x = 0; x < width; x++) {

			for (int y = 0; y < height; y++) {

				FreeImage_GetPixelColor(fib, x, y, &color);

				float blue = color.rgbBlue;
				float green = color.rgbGreen;
				float red = color.rgbRed;
				redChannel->push_back(red);
				greenChannel->push_back(green);
				blueChannel->push_back(blue);

			}

		}

		FreeImage_Unload(fib);
		FreeImage_DeInitialise();
	}

	std::vector<float> imageChannelNormalization(std::vector<float> *channel) {

		float maxColorChannel = *std::max_element(channel->begin(),
				channel->end());
		float minColorChannel = *std::min_element(channel->begin(),
				channel->end());

		std::vector<float> result;

		for (int i = 0; i < channel->size(); i++) {
			result.push_back(
					(channel->at(i) - minColorChannel)
							/ (maxColorChannel - minColorChannel));
		}

		channel->clear();

		return result;

	}
};

class KernelGenerator {

public:

	void initializeKernelUsingXavierAlgorithm(int kernelHeight,
			int kernelWeight, int channelNumber, std::vector<float> * kernel) {

		//随机数生成器初始化
		std::random_device rd;
		//使用马特赛特旋转演算法伪随机数生成器
		std::mt19937 generator(rd());

		float core = sqrt(3.0f / (kernelHeight * kernelWeight * channelNumber));

		std::uniform_real_distribution<> distribution(-core, core);

		for (int i = 0; i < kernel->size(); i++) {
			kernel->at(i) = static_cast<float>(distribution(generator));
		}

//		for(int i = 0 ; i < kernel->size();i++){
//			std::cout << kernel->at(i) << std::endl;
//		}

	}

};

//class CuNeuralNetwork {
//
//public:
//
//	//input
//	hipdnnHandle_t *cudnnHandle;
//	hipdnnTensorDescriptor_t * inputDataTensorDescriptor;
//	hipdnnFilterDescriptor_t * kernelDescriptor;
//	hipdnnConvolutionDescriptor_t * convolutionDescriptor;
//	hipdnnTensorDescriptor_t * outputDataTensorDescriptor;
//	int executeBatchSize;
//	int imageHeight;
//	int imageWidth;
//	int kernelHeight;
//	int kernelWidth;
//	int inputChannels;
//	int outputChannels;
//
//	//output
//	size_t workspaceSizeInByte;
//	hipdnnConvolutionFwdAlgo_t * algorithm;
//	int outputImageNumber;
//	int outputChannelsOfEachImage;
//	int outputFeaturemapHeight;
//	int outputFeaturemapWidth;
//
//public:
//
//	size_t initializeConvolutionalLayerTensorDescriptor(
//			hipdnnHandle_t *cudnnHandle,
//			hipdnnTensorDescriptor_t * inputDataTensorDescriptor,
//			hipdnnFilterDescriptor_t * kernelDescriptor,
//			hipdnnConvolutionDescriptor_t * convolutionDescriptor,
//			hipdnnTensorDescriptor_t * outputDataTensorDescriptor,
//			hipdnnConvolutionFwdAlgo_t * algorithm, int executeBatchSize,
//			int imageHeight, int imageWidth, int kernelHeight, int kernelWidth,
//			int inputChannels, int outputChannels) {
//
//		//输入数据设定
//		checkCUDNN(hipdnnCreateTensorDescriptor(inputDataTensorDescriptor));
//		checkCUDNN(
//				hipdnnSetTensor4dDescriptor(*inputDataTensorDescriptor,
//						HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, executeBatchSize,
//						inputChannels, imageHeight, imageWidth));
//
//		//卷积核设定
//		checkCUDNN(hipdnnCreateFilterDescriptor(kernelDescriptor));
//		checkCUDNN(
//				hipdnnSetFilter4dDescriptor(*kernelDescriptor, HIPDNN_DATA_FLOAT,
//						outputChannels, outputChannels, kernelHeight,
//						kernelWidth));
//
//		//卷积操作设定
//		checkCUDNN(hipdnnCreateConvolutionDescriptor(convolutionDescriptor));
//		//零填充的行数与列数：0 卷积的水平和垂直的滑动长度：1 x，y向上取样的比例尺：1
//		//不使用卷积操作，因为卷积操作要旋转卷积核，而互相关操作无需旋转卷积核
//		checkCUDNN(
//				hipdnnSetConvolution2dDescriptor(*convolutionDescriptor, 0, 0, 1,
//						1, 1, 1, HIPDNN_CROSS_CORRELATION));
//
//		//输出数据设定
//		//获取：图片数量，输出featuremap数量，featuremap的高度，featuremap的宽度
//
//		checkCUDNN(
//				hipdnnGetConvolution2dForwardOutputDim(*convolutionDescriptor,
//						*inputDataTensorDescriptor, *kernelDescriptor,
//						&this->imageNumber, &this->ChannelsOfImage,
//						&this->featuremapHeight, &this->featuremapWidth));
//
//		checkCUDNN(hipdnnCreateTensorDescriptor(outputDataTensorDescriptor));
//		checkCUDNN(
//				hipdnnSetTensor4dDescriptor(*outputDataTensorDescriptor,
//						HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
//						outputData->imageNumber, outputData->ChannelsOfImage,
//						outputData->featuremapHeight,
//						outputData->featuremapWidth));
//
//		//选择FP算法
//		checkCUDNN(
//				hipdnnGetConvolutionForwardAlgorithm(*cudnnHandle,
//						*inputDataTensorDescriptor, *kernelDescriptor,
//						*convolutionDescriptor, *outputDataTensorDescriptor,
//						HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, algorithm));
//
//		//获取workspace的大小
//		size_t workspaceSizeInByte = 0;
//		checkCUDNN(
//				hipdnnGetConvolutionForwardWorkspaceSize(*cudnnHandle,
//						*inputDataTensorDescriptor, *kernelDescriptor,
//						*convolutionDescriptor, *outputDataTensorDescriptor,
//						*algorithm, &workspaceSizeInByte));
//
//		return workspaceSizeInByte;
//
//	}
//
//};

class Utility {

public:
	static float* VectorToArray(std::vector<float> * input) {

		float * array;
		array = new float[input->size()];

		for (int i = 0; i < input->size(); i++) {
			array[i] = input->at(i);
		}

		return array;
	}
};

class TestCase {

public:
	static void TestCase1(float * data, float * kernel) {

		float sum = 0.0f;

		sum += data[0] * kernel[0];
		sum += data[1] * kernel[1];
		sum += data[2] * kernel[2];
		sum += data[540] * kernel[3];
		sum += data[541] * kernel[4];
		sum += data[542] * kernel[5];
		sum += data[1080] * kernel[6];
		sum += data[1081] * kernel[7];
		sum += data[1082] * kernel[8];

		std::cout << " Ground Truth : " << sum << std::endl;

	}

	static void printDynamicArray(float * array, int length) {
		for (int i = 0; i < length; i++) {
			std::cout << array[i] << std::endl;
		}
	}

};

int main() {

	int filePathMaxLength = 256;
	int imageHeight = 540;
	int imageWidth = 540;
	int kernelHeight = 3;
	int kernelWidth = 3;

	//读取图片到RGB三个通道
	char imagePath[filePathMaxLength];
	getcwd(imagePath, filePathMaxLength);
	strcat(imagePath, "/trainingset/");
	strcat(imagePath, "train1.png");

	std::vector<float> redChannel;
	std::vector<float> greenChannel;
	std::vector<float> blueChannel;

	ImageProcessor processor;
	processor.readRGBImage(imagePath, &redChannel, &greenChannel, &blueChannel);

	redChannel = processor.imageChannelNormalization(&redChannel);
	greenChannel = processor.imageChannelNormalization(&greenChannel);
	blueChannel = processor.imageChannelNormalization(&blueChannel);

	float * h_input_data;
	h_input_data = Utility::VectorToArray(&redChannel);
	//END

	//卷积核数据初始化
	KernelGenerator generator;
	std::vector<float> kernel(kernelHeight * kernelWidth);
	generator.initializeKernelUsingXavierAlgorithm(kernelHeight, kernelWidth, 1,
			&kernel);

	float * h_kernel;
	h_kernel = Utility::VectorToArray(&kernel);
	//END

	//GPU的查询与选择
	int GPUs;
	checkCudaErrors(hipGetDeviceCount(&GPUs));
	if (GPUs > 0) {
		checkCudaErrors(hipSetDevice(0));
	} else {
		return 0;
	}

	//cudnn初始化
	hipdnnHandle_t cudnnHandle = NULL;
	hipdnnCreate(&cudnnHandle);

	hipdnnTensorDescriptor_t inputDataTensor;
	hipdnnFilterDescriptor_t kernelDescriptor;
	hipdnnConvolutionDescriptor_t convolutionDescriptor;
	hipdnnTensorDescriptor_t outputDataTensor;
	hipdnnConvolutionFwdAlgo_t algorithm;
	int executeBatchSize = 1;
	int inputFeaturemaps = 1;
	int outputFeaturemaps = 1;
	size_t workspaceSizeInByte = 0;
	int outputImages;
	int outputFeaturemapsForEachImage;
	int outputFeaturemapHeight;
	int outputFeaturemapWidth;
	float d_input_data;
	float d_kernel;
	float d_output_data;
	void * d_cudnn_workspace = nullptr;

	CuNeuralNetwork network;

	network.initializeConvolutionalLayerTensorDescriptor(&cudnnHandle,
			&inputDataTensor, &kernelDescriptor, &convolutionDescriptor,
			&outputDataTensor, &algorithm, executeBatchSize, imageWidth,
			imageHeight, kernelHeight, kernelWidth, inputFeaturemaps,
			outputFeaturemaps, &workspaceSizeInByte, &outputImages,
			&outputFeaturemapsForEachImage, &outputFeaturemapHeight,
			&outputFeaturemapWidth);

	std::cout << outputImages << " " << outputFeaturemapsForEachImage << " "
			<< outputFeaturemapHeight << " " << outputFeaturemapWidth
			<< std::endl;

	std::cout <<inputDataTensor<< std::endl;

	network.syncTrainingDataToDevice(h_input_data, &d_input_data, h_kernel,
			&d_kernel, &d_output_data, d_cudnn_workspace, executeBatchSize,
			imageHeight, imageWidth, kernelHeight, kernelWidth,
			inputFeaturemaps, outputFeaturemaps, outputImages,
			outputFeaturemapsForEachImage, outputFeaturemapHeight,
			outputFeaturemapWidth, workspaceSizeInByte);

	float alpha = 1.0f, beta = 0.0f;

	checkCUDNN(
			hipdnnConvolutionForward(cudnnHandle, &alpha, inputDataTensor,
					&d_input_data, kernelDescriptor, &d_kernel,
					convolutionDescriptor, algorithm, d_cudnn_workspace,
					workspaceSizeInByte, &beta, outputDataTensor,
					&d_output_data));

//	float * h_output_data = new float[outputImages
//			* outputFeaturemapsForEachImage * outputFeaturemapHeight
//			* outputFeaturemapWidth];
//
//	//定义动态float数组
//	float * h_output_data = new float[outputN * outputC * outputH * outputH];
//
//	checkCudaErrors(
//			hipMemcpyAsync(h_output_data, &d_output_data,
//					sizeof(float) * outputImages * outputFeaturemapsForEachImage
//							* outputFeaturemapHeight * outputFeaturemapWidth,
//					hipMemcpyDeviceToHost));
//
//	checkCudaErrors(hipDeviceSynchronize());
//
//	TestCase::TestCase1(h_output_data, h_kernel);
//
//	std::cout << " test result : " << h_output_data[0] << std::endl;

//checkCUDNN(hipdnnDestroyTensorDescriptor(redChannelDataTensor));

}

