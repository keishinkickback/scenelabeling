/*
 ============================================================================
 Name        : scenelabeling.cu
 Author      : liujinhang @ whut
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <sstream>
#include <numeric>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>
#include <FreeImage.h>
#include <unistd.h>
#include <string.h>
#include <algorithm>

#include "CuNeuralNetwork.h"
#include "ImageProcessor.h"
#include "KernelGenerator.h"
#include "Utility.h"
#include "TestCase.h"

#include <vector>

//////////////////////////////////////////////////////////////////////////////
// Error handling
// Adapted from the CUDNN classification code
// sample: https://developer.nvidia.com/cuDNN

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

int main() {

	int filePathMaxLength = 256;
	int imageHeight = 540;
	int imageWidth = 540;
	int kernelHeight = 3;
	int kernelWidth = 3;
	//输入featuremap为1个，即图片原始数据。输出为1个featuremap，所以需要1个卷积核。
	int inputFeaturemaps = 1;
	int outputFeaturemaps = 1;

	//读取图片到RGB三个通道
	char imagePath[filePathMaxLength];
	getcwd(imagePath, filePathMaxLength);
	strcat(imagePath, "/trainingset/");
	strcat(imagePath, "train1.png");
	std::vector<float> redChannel;
	std::vector<float> greenChannel;
	std::vector<float> blueChannel;
	float * h_input_data;
	ImageProcessor processor;
	processor.readRGBImage(imagePath, &redChannel, &greenChannel, &blueChannel);
	redChannel = processor.imageChannelNormalization(&redChannel);
	greenChannel = processor.imageChannelNormalization(&greenChannel);
	blueChannel = processor.imageChannelNormalization(&blueChannel);
	h_input_data = Utility::VectorToArray(&redChannel);

	//卷积核初始化
	KernelGenerator generator;
	float * h_kernel;
	std::vector<float> kernel(kernelHeight * kernelWidth);
	generator.initializeKernelUsingXavierAlgorithm(kernelHeight, kernelWidth,
			outputFeaturemaps, &kernel);
	h_kernel = Utility::VectorToArray(&kernel);

	//偏置项初始化
	//与卷积核同样的方法，同样的数量
	float * h_bias;
	std::vector<float> bias(kernelHeight * kernelWidth);
	generator.initializeBiasUsingXavierAlgorithm(kernelHeight, kernelWidth,
			outputFeaturemaps, &bias);
	h_bias = Utility::VectorToArray(&bias);

	//GPU设定
	int GPUs;
	checkCudaErrors(hipGetDeviceCount(&GPUs));
	if (GPUs > 0) {
		checkCudaErrors(hipSetDevice(0));
	} else {
		return 0;
	}

	//cudnn初始化
	hipdnnHandle_t cudnnHandle = NULL;
	hipdnnCreate(&cudnnHandle);

	CuNeuralNetwork network;

	//输入数据设定
	hipdnnTensorDescriptor_t inputDataTensor;
	float * d_data = network.createInputDataLayer(h_input_data,
			&inputDataTensor, 1, 1, imageHeight, imageWidth);

	//卷积核设定
	hipdnnFilterDescriptor_t kernelDescriptor;
	float * d_kernel = network.createKernel(h_kernel, &kernelDescriptor,
			inputFeaturemaps, outputFeaturemaps, kernelHeight, kernelWidth);

	//卷积操作设定
	hipdnnConvolutionDescriptor_t convolutionDescriptor;
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolutionDescriptor));
	//零填充的行数与列数：0,卷积的水平和垂直的滑动长度：1,x，y向上取样的比例尺：1
	//不使用卷积操作，因为卷积操作要旋转卷积核，而这里不需要旋转，互相关就是无需旋转的卷积乘法。
	checkCUDNN(
			hipdnnSetConvolution2dDescriptor(convolutionDescriptor, 0, 0, 1, 1,
					1, 1, HIPDNN_CROSS_CORRELATION));

	//输出数据设定
	hipdnnTensorDescriptor_t outputDataTensor;
	OutputDim outputDim;
	float *d_output_data = network.createOutputDataLayer(&inputDataTensor,
			&kernelDescriptor, &convolutionDescriptor, &outputDataTensor,
			&outputDim);

	//选择FP算法
	hipdnnConvolutionFwdAlgo_t algorithm;
	checkCUDNN(
			hipdnnGetConvolutionForwardAlgorithm(cudnnHandle, inputDataTensor,
					kernelDescriptor, convolutionDescriptor, outputDataTensor,
					HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algorithm));

	//设置workspace的大小
	size_t workspaceSizeInByte = 0;
	checkCUDNN(
			hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
					inputDataTensor, kernelDescriptor, convolutionDescriptor,
					outputDataTensor, algorithm, &workspaceSizeInByte));
	void *d_cudnn_workspace = nullptr;
	checkCudaErrors(hipMalloc(&d_cudnn_workspace, workspaceSizeInByte));

	checkCudaErrors(hipDeviceSynchronize());

	//FP计算
	float alpha = 1.0f, beta = 0.0f;

	//卷积运算
	checkCUDNN(
			hipdnnConvolutionForward(cudnnHandle, &alpha, inputDataTensor,
					d_data, kernelDescriptor, d_kernel, convolutionDescriptor,
					algorithm, d_cudnn_workspace, workspaceSizeInByte, &beta,
					outputDataTensor, d_output_data));

	//偏置项设定
	hipdnnTensorDescriptor_t biasTensorDescriptor;
	float *d_bias = network.addBiasUnits(h_bias, &biasTensorDescriptor,
			outputFeaturemaps, kernelHeight, kernelWidth);

	//加上偏置项
	alpha = 1.0f, beta = 1.0f;
	checkCUDNN(
			hipdnnAddTensor(cudnnHandle, CUDNN_ADD_SAME_C, &alpha,
					biasTensorDescriptor, d_bias, &beta, outputDataTensor,
					d_output_data));

	//池化设定
	alpha = 1.0f, beta = 0.0f;
	hipdnnPoolingDescriptor_t poolingDescriptor;
	hipdnnTensorDescriptor_t poolingDataTensorDescriptor;
	OutputDim poolingOutputDim;
	int poolingWindowHeight = 3;
	int poolingWindowWidth = 3;
	int poolingVerticalStride = 1;
	int poolingHorizontalStride = 1;
	float * d_pooling_output_data = network.createPoolingLayer(d_output_data,
			&outputDataTensor, &poolingDescriptor, &poolingDataTensorDescriptor,
			&outputDim, poolingWindowHeight, poolingWindowWidth,
			poolingVerticalStride, poolingHorizontalStride, &poolingOutputDim);

	//池化运算
	checkCUDNN(
			hipdnnPoolingForward(cudnnHandle, poolingDescriptor, &alpha,
					outputDataTensor, d_output_data, &beta,
					poolingDataTensorDescriptor, d_pooling_output_data));

	//d_pooling_output_data数据回传
	float * h_pooling_output_data = new float[outputDim.outputImages
			* outputDim.outputFeaturemapsForEachImage
			* poolingOutputDim.outputFeaturemapHeight
			* poolingOutputDim.outputFeaturemapWidth];
	checkCudaErrors(
			hipMemcpyAsync(h_pooling_output_data, d_pooling_output_data,
					sizeof(float) * outputDim.outputImages
							* outputDim.outputFeaturemapsForEachImage
							* poolingOutputDim.outputFeaturemapHeight
							* poolingOutputDim.outputFeaturemapWidth,
					hipMemcpyDeviceToHost));

	//d_output_data数据回传
	float * h_output_data =
			new float[outputDim.outputImages
					* outputDim.outputFeaturemapsForEachImage
					* outputDim.outputFeaturemapHeight
					* outputDim.outputFeaturemapWidth];
	checkCudaErrors(
			hipMemcpyAsync(h_output_data, d_output_data,
					sizeof(float) * outputDim.outputImages
							* outputDim.outputFeaturemapsForEachImage
							* outputDim.outputFeaturemapHeight
							* outputDim.outputFeaturemapWidth,
					hipMemcpyDeviceToHost));

	//测试用例1
	TestCase::TestCase1(h_input_data, h_kernel, h_bias, h_output_data,
			h_pooling_output_data);

	//destroy section
	//checkCUDNN(hipdnnDestroyTensorDescriptor(redChannelDataTensor));

}
