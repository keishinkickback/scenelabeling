#include <iostream>
#include <sstream>
#include <numeric>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>
#include <FreeImage.h>
#include <unistd.h>
#include <string.h>
#include <algorithm>

#include "includes/CuTest.h"

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

//int main(void) {
//	float alpha = 1.0;
//	float beta = 0.0;
//	float h_A[6] = { 1, 1, 2, 2, 3, 3 };
//	float h_B[2] = { 1, 1 };
//	float h_C[3];
//	float *d_a, *d_b, *d_c;
//	checkCudaErrors(hipMalloc((void** ) &d_a, 6 * sizeof(float)));
//	checkCudaErrors(hipMalloc((void** ) &d_b, 2 * sizeof(float)));
//	checkCudaErrors(hipMalloc((void** ) &d_c, 3 * sizeof(float)));
//	checkCudaErrors(
//			hipMemcpy(d_a, &h_A, 6 * sizeof(float), hipMemcpyHostToDevice));
//	checkCudaErrors(
//			hipMemcpy(d_b, &h_B, 2 * sizeof(float), hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemset(d_c, 0, 3 * sizeof(float)));
//	hipblasHandle_t handle;
//	hipblasCreate(&handle);
//
//	CuNeuralNetwork network;
//
//	hipblasSgemm(
//			handle,
//			HIPBLAS_OP_N,
//			HIPBLAS_OP_N,
//			1,
//			3,
//			2,
//			&alpha,
//			d_b,
//			1,
//			d_a,
//			2,
//			&beta,
//			d_c,
//			1);
//
//
//	checkCudaErrors(
//			hipMemcpy(h_C, d_c, 3 * sizeof(float), hipMemcpyDeviceToHost));
//	for (int i = 0; i < 3; i++) {
//		printf("%f\n", h_C[i]);
//	}
//	printf("\n");
//	return 0;
//}
